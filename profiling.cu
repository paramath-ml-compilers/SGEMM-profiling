#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <runner.cuh>
#include <hipblas.h>

// Kernel12 template parameters:
//   @tparam BM The threadblock size for M dimension SMEM caching.
//   @tparam BN The threadblock size for N dimension SMEM caching.
//   @tparam BK The threadblock size for K dimension SMEM caching.
//   @tparam WM M dim of continuous tile computed by each warp
//   @tparam WN N dim of continuous tile computed by each warp
//   @tparam WMITER The number of subwarp tiling steps in M dimension.
//   @tparam WNITER The number of subwarp tiling steps in N dimension.
//   @tparam TM The per-thread tile size for M dimension.
//   @tparam TN The per-thread tile size for N dimension.

#define cudaCheck(error) \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(error) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

int main() {
    // Use fixed large dimensions and GEMM parameters
    int m = 4096, n = 4096, k = 4096;
    float alpha = 0.5f, beta = 3.0f;

    cudaCheck(hipSetDevice(0));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    if(hipblasCreate(&handle)) {
        std::cerr << "Failed to create cuBLAS handle." << std::endl;
        return EXIT_FAILURE;
    }

    // Allocate host matrices
    size_t sizeA = m * k, sizeB = k * n, sizeC = m * n;
    float *hA = (float*)malloc(sizeof(float) * sizeA);
    float *hB = (float*)malloc(sizeof(float) * sizeB);
    float *hC = (float*)malloc(sizeof(float) * sizeC);
    // Initialize with random data
    for (size_t i = 0; i < sizeA; i++) hA[i] = static_cast<float>(rand()) / RAND_MAX;
    for (size_t i = 0; i < sizeB; i++) hB[i] = static_cast<float>(rand()) / RAND_MAX;
    for (size_t i = 0; i < sizeC; i++) hC[i] = static_cast<float>(rand()) / RAND_MAX;

    // Allocate device matrices
    float *dA, *dB, *dC;
    cudaCheck(hipMalloc((void**)&dA, sizeof(float) * sizeA));
    cudaCheck(hipMalloc((void**)&dB, sizeof(float) * sizeB));
    cudaCheck(hipMalloc((void**)&dC, sizeof(float) * sizeC));

    cudaCheck(hipMemcpy(dA, hA, sizeof(float) * sizeA, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dB, hB, sizeof(float) * sizeB, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC, hC, sizeof(float) * sizeC, hipMemcpyHostToDevice));

    // Timing kernel12 (run_kernel with kernel_num 12)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    run_kernel(13, m, n, k, alpha, dA, dB, beta, dC, handle);
    cudaCheck(hipDeviceSynchronize());

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    double flops = 2.0 * m * n * k;
    double gflops = (flops / (milliseconds/1000.0)) * 1e-9;

    std::cout << "Kernel12 execution time: " << (milliseconds/1000.0) << " s" << std::endl;
    std::cout << "Performance: " << gflops << " GFLOPS" << std::endl;

    free(hA); free(hB); free(hC);
    hipFree(dA); hipFree(dB); hipFree(dC);
    hipblasDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}


